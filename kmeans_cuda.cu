#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Constants
#define TILE_WIDTH 16
#define TILE_HEIGHT 16
#define TDIM_Y 2
#define BLOCK_SIZE 256

/**
 * K-means clustering algorithm implementation based on:
 * "Speeding up k-Means algorithm by GPUs" by Li et al.
 *
 * This implementation contains two strategies:
 * 1. Register-based approach for low-dimensional data
 * 2. Shared memory approach for high-dimensional data
 */

// Structure to hold dataset information
typedef struct {
    float* data;
    int n;       // number of data points
    int d;       // dimensionality
    int k;       // number of clusters
    float* centroids;
    int* assignments;  // cluster assignment for each point
    int* counts;       // count of points in each cluster
    int max_iterations;
    float threshold;
} KMeansData;

// Utility functions
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Allocate memory for K-means data
void allocateMemory(KMeansData* data) {
    // Host memory
    data->centroids = (float*)malloc(data->k * data->d * sizeof(float));
    data->assignments = (int*)malloc(data->n * sizeof(int));
    data->counts = (int*)malloc(data->k * sizeof(int));
    
    if (!data->centroids || !data->assignments || !data->counts) {
        fprintf(stderr, "Error: Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
}

// Initialize centroids randomly from data points
void initializeCentroids(KMeansData* data) {
    // Use the first k data points as initial centroids
    // In a real implementation, you might want to use k-means++ or random selection
    for (int i = 0; i < data->k; i++) {
        for (int j = 0; j < data->d; j++) {
            data->centroids[i * data->d + j] = data->data[i * data->d + j];
        }
    }
}

// Free allocated memory
void freeMemory(KMeansData* data, float* d_data, float* d_centroids, int* d_assignments) {
    free(data->centroids);
    free(data->assignments);
    free(data->counts);
    
    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_assignments);
}

/***************** Low-Dimensional Strategy (Register-Based) ******************/

// CUDA kernel for finding the closest centroid (register-based for low dimensional data)
__global__ void findClosestCentroidLowDim(float* data, float* centroids, int* assignments, 
                                       int n, int d, int k) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        float min_dist = FLT_MAX;
        int closest = 0;
        
        // Load data point into registers
        float point[16];  // Assuming max dimension of 16 for registers
        for (int j = 0; j < d; j++) {
            point[j] = data[tid * d + j];
        }
        
        // Find closest centroid
        for (int c = 0; c < k; c++) {
            float dist = 0.0f;
            
            // Calculate Euclidean distance
            for (int j = 0; j < d; j++) {
                float diff = point[j] - centroids[c * d + j];
                dist += diff * diff;
            }
            
            if (dist < min_dist) {
                min_dist = dist;
                closest = c;
            }
        }
        
        assignments[tid] = closest;
    }
}

/***************** High-Dimensional Strategy (Shared Memory) ******************/

// CUDA kernel for finding the closest centroid (shared memory-based for high dimensional data)
__global__ void findClosestCentroidHighDim(float* data, float* centroids, int* assignments,
                                        int n, int d, int k) {
    __shared__ float SMData[TILE_WIDTH][TILE_HEIGHT];
    
    // Calculate indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Calculate global indices
    int indexD = data + (by * TILE_HEIGHT * d) + (ty * d) + tx;
    int indexC = centroids + (bx * TILE_WIDTH) + (ty * blockDim.x) + tx;
    int indexR = assignments + (by * TILE_HEIGHT * k) + (bx * TILE_WIDTH) + (ty * blockDim.x) + tx;
    
    // Temporary result stored in registers
    float TResult[TILE_WIDTH];
    for (int i = 0; i < TILE_WIDTH; i++) {
        TResult[i] = 0.0f;
    }
    
    // Process data in tiles
    float* Alast = indexD + d;
    while (indexD < Alast) {
        // Load tile of data into shared memory
        if (tx < d && (by * TILE_HEIGHT + ty) < n) {
            SMData[tx][ty] = *indexD;
        }
        __syncthreads();
        
        // Compute distance for this tile
        for (int i = 0; i < TILE_WIDTH && i < d; i++) {
            if ((by * TILE_HEIGHT + ty) < n && (bx * TILE_WIDTH + tx) < k) {
                float diff = SMData[i][ty] - centroids[bx * TILE_WIDTH + tx + i * k];
                TResult[tx] += diff * diff;
            }
        }
        
        indexD += TILE_WIDTH;
        __syncthreads();
    }
    
    // Find minimum distance and assign cluster
    if ((by * TILE_HEIGHT + ty) < n && (bx * TILE_WIDTH + tx) < k) {
        float minDist = TResult[0];
        int minIndex = 0;
        
        for (int i = 1; i < TILE_WIDTH && i < k; i++) {
            if (TResult[i] < minDist) {
                minDist = TResult[i];
                minIndex = i;
            }
        }
        
        assignments[by * TILE_HEIGHT + ty] = bx * TILE_WIDTH + minIndex;
    }
}

// CUDA kernel to update centroids
__global__ void computeNewCentroidsKernel(float* data, int* assignments, float* centroids, 
                                       int* counts, int n, int d, int k) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < k * d) {
        int centroid_idx = tid / d;
        int dim = tid % d;
        
        float sum = 0.0f;
        int count = 0;
        
        for (int i = 0; i < n; i++) {
            if (assignments[i] == centroid_idx) {
                sum += data[i * d + dim];
                count++;
            }
        }
        
        if (count > 0) {
            centroids[tid] = sum / count;
        }
        
        if (dim == 0) {
            counts[centroid_idx] = count;
        }
    }
}

// Compute new centroids on CPU
void computeNewCentroidsCPU(KMeansData* data, float* d_data, int* d_assignments, float* d_centroids) {
    // Copy assignments back to host
    hipMemcpy(data->assignments, d_assignments, data->n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Reset counts and centroids
    for (int i = 0; i < data->k; i++) {
        data->counts[i] = 0;
        for (int j = 0; j < data->d; j++) {
            data->centroids[i * data->d + j] = 0.0f;
        }
    }
    
    // Sum up all points assigned to each centroid
    for (int i = 0; i < data->n; i++) {
        int cluster = data->assignments[i];
        data->counts[cluster]++;
        
        for (int j = 0; j < data->d; j++) {
            data->centroids[cluster * data->d + j] += data->data[i * data->d + j];
        }
    }
    
    // Divide by count to get means
    for (int i = 0; i < data->k; i++) {
        if (data->counts[i] > 0) {
            for (int j = 0; j < data->d; j++) {
                data->centroids[i * data->d + j] /= data->counts[i];
            }
        }
    }
    
    // Copy updated centroids back to device
    hipMemcpy(d_centroids, data->centroids, data->k * data->d * sizeof(float), hipMemcpyHostToDevice);
}

// Check for convergence
bool hasConverged(float* old_centroids, float* new_centroids, int k, int d, float threshold) {
    for (int i = 0; i < k; i++) {
        float distance = 0.0f;
        for (int j = 0; j < d; j++) {
            float diff = old_centroids[i * d + j] - new_centroids[i * d + j];
            distance += diff * diff;
        }
        distance = sqrt(distance);
        if (distance > threshold) {
            return false;
        }
    }
    return true;
}

// Main K-means function for low-dimensional data
void kmeansLowDim(KMeansData* data) {
    float *d_data, *d_centroids;
    int *d_assignments;
    
    // Allocate device memory
    hipMalloc((void**)&d_data, data->n * data->d * sizeof(float));
    hipMalloc((void**)&d_centroids, data->k * data->d * sizeof(float));
    hipMalloc((void**)&d_assignments, data->n * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_data, data->data, data->n * data->d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, data->centroids, data->k * data->d * sizeof(float), hipMemcpyHostToDevice);
    
    // Previous centroids for convergence check
    float* old_centroids = (float*)malloc(data->k * data->d * sizeof(float));
    
    // Configure kernel
    int blockSize = BLOCK_SIZE;
    int gridSize = (data->n + blockSize - 1) / blockSize;
    
    // Main loop
    for (int iter = 0; iter < data->max_iterations; iter++) {
        // Save current centroids for convergence check
        memcpy(old_centroids, data->centroids, data->k * data->d * sizeof(float));
        
        // Find closest centroid for each data point
        findClosestCentroidLowDim<<<gridSize, blockSize>>>(d_data, d_centroids, d_assignments, 
                                                       data->n, data->d, data->k);
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), "Kernel execution failed");
        
        // Compute new centroids
        computeNewCentroidsCPU(data, d_data, d_assignments, d_centroids);
        
        // Check for convergence
        if (hasConverged(old_centroids, data->centroids, data->k, data->d, data->threshold)) {
            printf("Converged after %d iterations\n", iter + 1);
            break;
        }
    }
    
    // Copy final assignments back to host
    hipMemcpy(data->assignments, d_assignments, data->n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Clean up
    free(old_centroids);
    freeMemory(data, d_data, d_centroids, d_assignments);
}

// Main K-means function for high-dimensional data
void kmeansHighDim(KMeansData* data) {
    float *d_data, *d_centroids;
    int *d_assignments;
    
    // Allocate device memory
    hipMalloc((void**)&d_data, data->n * data->d * sizeof(float));
    hipMalloc((void**)&d_centroids, data->k * data->d * sizeof(float));
    hipMalloc((void**)&d_assignments, data->n * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_data, data->data, data->n * data->d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, data->centroids, data->k * data->d * sizeof(float), hipMemcpyHostToDevice);
    
    // Previous centroids for convergence check
    float* old_centroids = (float*)malloc(data->k * data->d * sizeof(float));
    
    // Configure kernel for high-dimensional strategy
    dim3 dimBlock(TILE_WIDTH, TDIM_Y);
    dim3 dimGrid((data->k + TILE_WIDTH - 1) / TILE_WIDTH, 
                (data->n + TILE_HEIGHT - 1) / TILE_HEIGHT);
    
    // Main loop
    for (int iter = 0; iter < data->max_iterations; iter++) {
        // Save current centroids for convergence check
        memcpy(old_centroids, data->centroids, data->k * data->d * sizeof(float));
        
        // Find closest centroid for each data point using shared memory strategy
        findClosestCentroidHighDim<<<dimGrid, dimBlock>>>(d_data, d_centroids, d_assignments, 
                                                       data->n, data->d, data->k);
        hipDeviceSynchronize();
        checkCudaError(hipGetLastError(), "Kernel execution failed");
        
        // Compute new centroids
        computeNewCentroidsCPU(data, d_data, d_assignments, d_centroids);
        
        // Check for convergence
        if (hasConverged(old_centroids, data->centroids, data->k, data->d, data->threshold)) {
            printf("Converged after %d iterations\n", iter + 1);
            break;
        }
    }
    
    // Copy final assignments back to host
    hipMemcpy(data->assignments, d_assignments, data->n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Clean up
    free(old_centroids);
    freeMemory(data, d_data, d_centroids, d_assignments);
}

// Main entry point
int main(int argc, char** argv) {
    // Check for command line arguments
    if (argc < 5) {
        printf("Usage: %s <data_file> <n> <d> <k> [max_iterations] [threshold]\n", argv[0]);
        return 1;
    }
    
    // Parse command line arguments
    char* filename = argv[1];
    int n = atoi(argv[2]);  // number of data points
    int d = atoi(argv[3]);  // dimensionality
    int k = atoi(argv[4]);  // number of clusters
    int max_iterations = (argc > 5) ? atoi(argv[5]) : 100;
    float threshold = (argc > 6) ? atof(argv[6]) : 1e-4;
    
    // Initialize k-means data
    KMeansData kmeans_data;
    kmeans_data.n = n;
    kmeans_data.d = d;
    kmeans_data.k = k;
    kmeans_data.max_iterations = max_iterations;
    kmeans_data.threshold = threshold;
    
    // Allocate memory for data
    kmeans_data.data = (float*)malloc(n * d * sizeof(float));
    if (!kmeans_data.data) {
        fprintf(stderr, "Error: Memory allocation failed for data\n");
        return 1;
    }
    
    // Read data from file
    FILE* file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error: Could not open file %s\n", filename);
        free(kmeans_data.data);
        return 1;
    }
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < d; j++) {
            if (fscanf(file, "%f", &kmeans_data.data[i * d + j]) != 1) {
                fprintf(stderr, "Error: Invalid data format\n");
                fclose(file);
                free(kmeans_data.data);
                return 1;
            }
        }
    }
    fclose(file);
    
    // Allocate memory for results
    allocateMemory(&kmeans_data);
    
    // Initialize centroids
    initializeCentroids(&kmeans_data);
    
    // Start timer
    clock_t start = clock();
    
    // Run k-means algorithm based on dimensionality
    if (d <= 16) { // Low-dimensional strategy using registers
        printf("Using low-dimensional strategy (register-based)\n");
        kmeansLowDim(&kmeans_data);
    } else { // High-dimensional strategy using shared memory
        printf("Using high-dimensional strategy (shared memory)\n");
        kmeansHighDim(&kmeans_data);
    }
    
    // End timer
    clock_t end = clock();
    double elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    
    printf("K-means completed in %.3f seconds\n", elapsed);
    
    // Output results
    printf("Final centroids:\n");
    for (int i = 0; i < k; i++) {
        printf("Centroid %d: ", i);
        for (int j = 0; j < d; j++) {
            printf("%.4f ", kmeans_data.centroids[i * d + j]);
        }
        printf("\n");
    }
    
    // Cleanup
    free(kmeans_data.data);
    free(kmeans_data.centroids);
    free(kmeans_data.assignments);
    free(kmeans_data.counts);
    
    return 0;
}